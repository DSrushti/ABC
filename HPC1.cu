#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<time.h>
void intarr(int* arr,int N);
void printarr(int* arr,int N);
__global__ 
void cal(int *arr,int N)
{
    int tid = threadIdx.x;
    int no_threads = blockDim.x;
    //printf("tid %d\n",tid);
    //printf("no %d\n",no_threads);
    int step =1;
    while(no_threads>0)
    {
        
        //printf("tid %d\n",tid);
        if(tid<no_threads)
        {
            
            int f = tid*step*2;
            int s = f + step;
            arr[f] += arr[s];
        }
        no_threads>>=1;
        step<<=1;
    }
}
__global__
void maxcal(int *arr,int N)
{
    int tid = threadIdx.x;
    int no_threads = blockDim.x;
    //printf("tid %d\n",tid);
    //printf("no %d\n",no_threads);
    int step =1;
    while(no_threads>0)
    {
        
        //printf("tid %d\n",tid);
        if(tid<no_threads)
        {
            
            int f = tid*step*2;
            int s = f + step;
            if(arr[f]<arr[s])
              arr[f] = arr[s];
        }
        no_threads>>=1;
        step<<=1;
    }
}

__global__
void stdcal(int *arr,int N,int avg)
{
    int tid = threadIdx.x;
    int no_threads = blockDim.x;
    //printf("tid %d\n",tid);
    //printf("no %d\n",no_threads);
    int step =1;
    int f = tid*step*2;
    int s = f + step;
    arr[f] = (arr[f] - avg)*(arr[f] - avg);
    arr[s] = (arr[s] - avg)*(arr[s] - avg);
}


int main()
{
    srand(time(NULL));
    int* a;
    int* d_a;
    const int N = 4;
    const int size = sizeof(a)*N;
    a = (int *)malloc(size);
    intarr(a,N);
    printf("Initial\n");
    printarr(a,N);
    hipMalloc(&d_a,size);
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    cal<<<1,N/2>>>(d_a,N);
    hipMemcpy(a,d_a,size,hipMemcpyDeviceToHost);
    int sum = a[0];    
    printf("Final sum %d\n",a[0]);
    printf("Final avg %d\n",sum/N);
    //maxcal<<<1,N/2>>>(d_a,N);
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    stdcal<<<1,N/2>>>(d_a,N,sum/N);
    hipMemcpy(a,d_a,size,hipMemcpyDeviceToHost);
    printf("Final\n");
    printarr(a,N);
}

void intarr(int* arr,int N)
{
 for(int i=0;i<N;i++)
    {
        arr[i] = rand()%N;
    }
}
void printarr(int* arr,int N)
{
    for(int i=0;i<N;i++)
    {
        printf("%d \n",arr[i]);
    }
}
